// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/cuda_graph_exec.hpp>

#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

CUDAGraphExec::CUDAGraphExec(ihipGraph* graph) {
  check_error << hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
}

CUDAGraphExec::~CUDAGraphExec() {
  check_error << hipGraphExecDestroy(instance);
}

void CUDAGraphExec::launch(ihipStream_t* stream) {
  check_error << hipGraphLaunch(instance, stream);
}

}  // namespace gtsam_points