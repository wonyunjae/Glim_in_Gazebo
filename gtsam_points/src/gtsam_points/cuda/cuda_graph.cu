// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/cuda_graph.cuh>

namespace gtsam_points {

CUDAGraph::CUDAGraph() {
  check_error << hipGraphCreate(&graph, 0);
}

CUDAGraph::~CUDAGraph() {
  check_error << hipGraphDestroy(graph);
}

void CUDAGraph::add_dependency(hipGraphNode* from, hipGraphNode* to) {
  check_error << hipGraphAddDependencies(graph, &from, &to, 1);
}

std::shared_ptr<CUDAGraphExec> CUDAGraph::instantiate() {
  return std::make_shared<CUDAGraphExec>(graph);
}

}  // namespace gtsam_points
