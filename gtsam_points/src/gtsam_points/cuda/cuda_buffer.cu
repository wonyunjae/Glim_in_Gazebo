// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/cuda_buffer.hpp>

#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

CUDABuffer::CUDABuffer(bool use_pinned_buffer) : use_pinned_buffer(use_pinned_buffer), buffer_size(0), h_buffer(nullptr), d_buffer(nullptr) {}

CUDABuffer::~CUDABuffer() {
  if (h_buffer) {
    check_error << hipHostFree(h_buffer);
  }
  if (d_buffer) {
    check_error << hipFreeAsync(d_buffer, 0);
  }
}

void CUDABuffer::resize(size_t size, ihipStream_t* stream) {
  if (buffer_size < size) {
    size = size * 1.2;
    if (use_pinned_buffer) {
      check_error << hipHostFree(h_buffer);
      check_error << hipHostMalloc(&h_buffer, size);
    }

    check_error << hipFreeAsync(d_buffer, stream);
    check_error << hipMallocAsync(&d_buffer, size, stream);
    buffer_size = size;
  }
}

void CUDABuffer::upload(size_t size, ihipStream_t* stream) {
  if (size > buffer_size) {
    std::cerr << "error: data size must be smaller than buffer_size!!" << std::endl;
    std::cerr << "     : size=" << size << " buffer_size=" << buffer_size << std::endl;
    abort();
  }

  if (!use_pinned_buffer) {
    std::cerr << "error: pinned buffer must be enabled!!" << std::endl;
    abort();
  }

  check_error << hipMemcpyAsync(d_buffer, h_buffer, size, hipMemcpyHostToDevice, stream);
}

void CUDABuffer::upload(ihipStream_t* stream) {
  upload(buffer_size, stream);
}

void CUDABuffer::upload(const void* buffer, size_t size, ihipStream_t* stream) {
  resize(size, stream);

  const void* src_buffer = buffer;
  if (use_pinned_buffer) {
    check_error << hipMemcpyAsync(h_buffer, buffer, size, hipMemcpyHostToHost, stream);
    src_buffer = h_buffer;
  }

  check_error << hipMemcpyAsync(d_buffer, src_buffer, size, hipMemcpyHostToDevice, stream);
}

void CUDABuffer::download(ihipStream_t* stream) {
  if (!use_pinned_buffer) {
    std::cerr << "error: trying to dowload data to disabled pinned host buffer!!" << std::endl;
    abort();
  }

  check_error << hipMemcpyAsync(h_buffer, d_buffer, buffer_size, hipMemcpyDeviceToHost, stream);
}

void CUDABuffer::download(void* buffer, size_t size, ihipStream_t* stream) {
  if (use_pinned_buffer) {
    check_error << hipMemcpyAsync(h_buffer, d_buffer, buffer_size, hipMemcpyDeviceToHost, stream);
    check_error << hipMemcpyAsync(buffer, h_buffer, size, hipMemcpyHostToHost, stream);
  } else {
    check_error << hipMemcpyAsync(buffer, d_buffer, size, hipMemcpyDeviceToHost, stream);
  }
  check_error << hipStreamSynchronize(stream);
}

size_t CUDABuffer::size() const {
  return buffer_size;
}

void* CUDABuffer::host_buffer() {
  return h_buffer;
}

void* CUDABuffer::device_buffer() {
  return d_buffer;
}

}  // namespace gtsam_points
