// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/cuda_device_prop.hpp>

namespace gtsam_points {

std::vector<std::string> cuda_device_names() {
  int num_devices = 0;
  hipGetDeviceCount(&num_devices);

  std::vector<std::string> names(num_devices);
  for (int i = 0; i < num_devices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    names[i] = prop.name;
  }

  return names;
}

}  // namespace gtsam_points
