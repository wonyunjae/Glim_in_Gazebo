// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

void CUDACheckError::operator<<(hipError_t error) const {
  if (error == hipSuccess) {
    return;
  }

  const std::string error_name = hipGetErrorName(error);
  const std::string error_string = hipGetErrorString(error);

  std::cerr << "warning: " << error_name << std::endl;
  std::cerr << "       : " << error_string << std::endl;
}

CUDACheckError check_error;

}  // namespace gtsam_points