// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)
#include <gtsam_points/cuda/cuda_memory.hpp>

#include <hip/hip_runtime.h>
#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

void* cuda_malloc(size_t size, ihipStream_t* stream) {
  void* ptr = nullptr;
  check_error << hipMallocAsync(&ptr, size, stream);
  return ptr;
}

void* cuda_malloc_and_upload(const void* data, size_t size, ihipStream_t* stream) {
  void* ptr = nullptr;
  check_error << hipMallocAsync(&ptr, size, stream);
  check_error << hipMemcpyAsync(ptr, data, size, hipMemcpyHostToDevice, stream);
  return ptr;
}

void cuda_free(void* ptr, ihipStream_t* stream) {
  check_error << hipFreeAsync(ptr, stream);
}

void cuda_host_to_device(void* dst, const void* src, size_t size, ihipStream_t* stream) {
  check_error << hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}

void cuda_device_to_host(void* dst, const void* src, size_t size, ihipStream_t* stream) {
  check_error << hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, stream);
}

void cuda_mem_get_info(size_t* free, size_t* total) {
  check_error << hipMemGetInfo(free, total);
}

}  // namespace gtsam_points
