// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/check_error_curand.cuh>

#include <hiprand.h>

namespace gtsam_points {

std::string curandGetErrorName(int error) {
  switch (error) {
    default:
      return "CURAND_UNKNOWN_ERROR";
    case HIPRAND_STATUS_SUCCESS:
      return "HIPRAND_STATUS_SUCCESS";
    case HIPRAND_STATUS_VERSION_MISMATCH:
      return "HIPRAND_STATUS_VERSION_MISMATCH";
    case HIPRAND_STATUS_NOT_INITIALIZED:
      return "HIPRAND_STATUS_NOT_INITIALIZED";
    case HIPRAND_STATUS_ALLOCATION_FAILED:
      return "HIPRAND_STATUS_ALLOCATION_FAILED";
    case HIPRAND_STATUS_TYPE_ERROR:
      return "HIPRAND_STATUS_TYPE_ERROR";
    case HIPRAND_STATUS_OUT_OF_RANGE:
      return "HIPRAND_STATUS_OUT_OF_RANGE";
    case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
      return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
    case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
      return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
    case HIPRAND_STATUS_LAUNCH_FAILURE:
      return "HIPRAND_STATUS_LAUNCH_FAILURE";
    case HIPRAND_STATUS_PREEXISTING_FAILURE:
      return "HIPRAND_STATUS_PREEXISTING_FAILURE";
    case HIPRAND_STATUS_INITIALIZATION_FAILED:
      return "HIPRAND_STATUS_INITIALIZATION_FAILED";
    case HIPRAND_STATUS_ARCH_MISMATCH:
      return "HIPRAND_STATUS_ARCH_MISMATCH";
    case HIPRAND_STATUS_INTERNAL_ERROR:
      return "HIPRAND_STATUS_INTERNAL_ERROR";
  }
}

void CurandCheckError::operator<<(int error) const {
  if (error == HIPRAND_STATUS_SUCCESS) {
    return;
  }

  const std::string error_name = curandGetErrorName(error);
  std::cerr << "warning: " << error_name << std::endl;
}

CurandCheckError check_curand;

}  // namespace gtsam_points