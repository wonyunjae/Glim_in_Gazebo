// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/factors/integrated_vgicp_derivatives.cuh>

#include <iostream>
#include <thrust/remove.h>
#include <thrust/iterator/transform_iterator.h>

#include <cub/device/device_reduce.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include <gtsam_points/cuda/kernels/pose.cuh>
#include <gtsam_points/cuda/kernels/untie.cuh>
#include <gtsam_points/cuda/kernels/lookup_voxels.cuh>
#include <gtsam_points/cuda/kernels/linearized_system.cuh>
#include <gtsam_points/cuda/kernels/vgicp_derivatives.cuh>
#include <gtsam_points/cuda/stream_temp_buffer_roundrobin.hpp>

#include <gtsam_points/types/gaussian_voxelmap_gpu.hpp>

namespace gtsam_points {

template <bool enable_surface_validation_>
void IntegratedVGICPDerivatives::issue_linearize_impl(const Eigen::Isometry3f* d_x, LinearizedSystem6* d_output) {
  //
  lookup_voxels_kernel<enable_surface_validation_> corr_kernel(*target, source->points_gpu, source->normals_gpu, d_x);
  hipcub::TransformInputIterator<thrust::pair<int, int>, lookup_voxels_kernel<enable_surface_validation_>, int*> corr_first(source_inliers, corr_kernel);

  vgicp_derivatives_kernel deriv_kernel(d_x, *target, source->points_gpu, source->covs_gpu);
  hipcub::TransformInputIterator<LinearizedSystem6, vgicp_derivatives_kernel, decltype(corr_first)> first(corr_first, deriv_kernel);

  void* temp_storage = nullptr;
  size_t temp_storage_bytes = 0;

  hipcub::DeviceReduce::Reduce(
    temp_storage,
    temp_storage_bytes,
    first,
    d_output,
    num_inliers,
    thrust::plus<LinearizedSystem6>(),
    LinearizedSystem6::zero(),
    stream);

  temp_storage = temp_buffer->get_buffer(temp_storage_bytes);

  hipcub::DeviceReduce::Reduce(
    temp_storage,
    temp_storage_bytes,
    first,
    d_output,
    num_inliers,
    thrust::plus<LinearizedSystem6>(),
    LinearizedSystem6::zero(),
    stream);
}

template void IntegratedVGICPDerivatives::issue_linearize_impl<true>(const Eigen::Isometry3f* d_x, LinearizedSystem6* d_output);
template void IntegratedVGICPDerivatives::issue_linearize_impl<false>(const Eigen::Isometry3f* d_x, LinearizedSystem6* d_output);

}  // namespace gtsam_points